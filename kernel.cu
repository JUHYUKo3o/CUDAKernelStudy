
#include <hip/hip_runtime.h>
__global__ void conv(const float *A,const float *K,const float *bias,int aw,int ah, int kw,int kh,float *C){
    /* A : input data */
    /* B : kernal     */
    int tx = threadIdx.x+blockIdx.x*blockDim.x;
    int ty = threadIdx.y+blockIdx.y*blockDim.y;

    int cw = blockDim.x*gridDim.x;

    int i = blockIdx.z % kw;
    int j = blockIdx.z / kw;

    /* index  */
    int a_idx = tx + ty*aw + (i+j*aw);
    int k_idx = i+j*kw;
    int c_idx = tx + (ty*cw);
    
    /* Convolution */
    C[c_idx] += A[a_idx]*K[k_idx]+bias[0]/(kw*kh);
}